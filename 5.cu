#include "hip/hip_runtime.h"
﻿#include <assert.h>
#include "hip/hip_runtime.h"
#include ""
#include "omp.h"
#include <hip/hip_runtime_api.h>
#include <string>
#include <iostream>
using namespace std;

#define typeof_memo int8_t
//ここでGPUメモリ(ホストメモリも)の使用量を調整
#define memory_usage_limit 1024 * 4 //MB
#define max_notes 4444

#define cudaDo_Check(err)\
{\
    auto tmp = err;\
    if (tmp)\
    {\
        cerr <<"!!!ERROR!!! " << endl;\
        cerr << "hipError_t: " << tmp << " " << hipGetErrorString(tmp) << " At " << __FILE__ << ":" << __LINE__ << endl;\
        hipDeviceReset();\
        return -1;\
    }\
}
#define kernelCheck() cudaDo_Check(hipGetLastError())

#define mul_elem(bl) (bl.x * bl.y * bl.z)

#define get_info()\
{\
    hipDeviceProp_t prop;\
    hipGetDeviceProperties(&prop, 0);\
    cout << "Device name: " << prop.name << endl;\
    cout << "Memory: " << prop.totalGlobalMem / 1024 / 1024 << "MB" << endl;\
    cout << "Threads per block: " << prop.maxThreadsPerBlock << endl;\
    cout << "Shared memory per block: " << prop.sharedMemPerBlock / 1024 << "KB" << endl;\
    cout << "Registers per block: " << prop.regsPerBlock << endl;\
    cout << "Warp size: " << prop.warpSize << endl;\
    cout << "Max threads per multiprocessor: " << prop.maxThreadsPerMultiProcessor << endl;\
    cout << "Max threads dim: " << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << endl;\
    cout << "Max grid size: " << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << endl;\
    cout << "Const memory: " << prop.totalConstMem / 1024 << "KB" << endl;\
}

__global__ void calc_score(int jc, int j, typeof_memo *memo, typeof_memo *points)
{
    int justice = j + blockIdx.x, a = blockIdx.y, m = blockIdx.z * mul_elem(blockDim) + threadIdx.x; //mだけグリッドzとブロック内で分けてる
    size_t idx = blockIdx.x * gridDim.y * gridDim.z * mul_elem(blockDim)
    + a * gridDim.z * mul_elem(blockDim)
    + m;
    if (max_notes < jc + justice + a + m || !(jc + justice + a + m)) // 0除算, 無駄な計算をかいひ
    {
        points[idx] = 0;
        return;
    }
    points[idx] = memo[(int)((jc*1.01f + justice + a * 0.5f) * 1000000 / (jc + justice + a + m))]
    + memo[jc] + memo[justice] + memo[a] + memo[m];
    return;
}

int main()
{
        get_info();
        dim3 block(32);

        //0~max_notesなので全部max_notes+1にしてる
        //mはブロック内でも複数やるからブロック内のスレッド数で除算, あまりが出たら+1
        int m_num = (max_notes+1)/mul_elem(block) + !!((max_notes+1)%mul_elem(block));
        int j_range = 1;
        while ((double)sizeof(typeof_memo) * j_range * (max_notes+1) * m_num * mul_elem(block)/1024/1024 < memory_usage_limit) j_range++;
        j_range-=1;

        dim3 grid(j_range, max_notes+1, m_num); //x: j(長さ), y: a, z: m
        int current_max = 1, score;
        typeof_memo *memo, *h_memo = new typeof_memo[1010000 + 1], *points, *h_points = new typeof_memo[mul_elem(grid) * mul_elem(block)];
        //cudaDo_Check(hipMallocManaged(&points, sizeof(typeof_memo) * mul_elem(grid) * mul_elem(block)));
        cudaDo_Check(hipMalloc(&points, sizeof(typeof_memo) * mul_elem(grid) * mul_elem(block)));
        cudaDo_Check(hipMalloc(&memo, sizeof(typeof_memo) * (1010000 + 1))); //よく使うからこれはグローバルメモリに載せる コンスタントにはでかすぎ
        #pragma omp parallel for //ほんとは各桁いい感じに回せばいいけどめんどいからゴリ押す
        for (int i = 0; i <= 1010000; i++)
        {
            h_memo[i] = 0;
            auto s_i = to_string(i);
            for (int j = 0; j < s_i.size(); j++) if (s_i[j] == '7') h_memo[i]++;
        }
        cudaDo_Check(hipMemcpy(memo, h_memo, sizeof(typeof_memo) * (1010000 + 1), hipMemcpyHostToDevice));
        delete[] h_memo;

        cout << "--------" << current_max << "--------" << endl;
        for (int jc = 0; jc <= max_notes; jc++)
        {
            for (int j = 0; j <= max_notes-jc; j+=j_range) //全部+αが探索されるから余りを考える必要はない
            {
                //cout << "Running calc_score kernel with jc=" << jc << ", j=" << j<< "~" << j+j_range-1 << endl;
                calc_score << <grid, block >> > (jc, j, memo, points);
                hipDeviceSynchronize();
                kernelCheck();
                cudaDo_Check(hipMemcpy(h_points, points, sizeof(typeof_memo) * mul_elem(grid) * mul_elem(block), hipMemcpyDeviceToHost));
                for (int jdiff = 0; jdiff < j_range; jdiff++) for (int attack = 0; attack <= max_notes-jc-(j+jdiff); attack++)
                for (int miss = 0; miss <= max_notes-jc-(j+jdiff)-attack; miss++)
                {
                        size_t idx = jdiff * grid.y * grid.z * mul_elem(block)
                        + attack * grid.z * mul_elem(block)
                        + miss;
                        if (current_max <= h_points[idx])
                        {
                            if(current_max < h_points[idx])
                            {
                                current_max = h_points[idx];
                                cout << "--------" << current_max << "--------" << endl;
                            }
                            score = (jc*1.01f + j +jdiff+ attack * 0.5f) * 1000000 / (jc + j + jdiff + attack + miss);
                            cout << jc+j+jdiff+attack+miss << " " << score << " "
                            << jc << "-" << j+jdiff << "-" << attack << "-" << miss << " " << +h_points[idx] << " 7(s)" << endl;
                        }
                }
            }
        }
    cout << "Exploration finished!" << endl;
    return 0;
}
