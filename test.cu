#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "omp.h"
#include <hip/hip_runtime_api.h>
#include <string>
#include <iostream>
using namespace std;
#define threads_per_block 32
#define typeof_memo int

__global__ void calc_score(int jc, int j, int max_notes, typeof_memo *memo, typeof_memo *points) {
    int a = blockIdx.y, m = blockIdx.z*threads_per_block + threadIdx.x; //mだけグリッドzとブロック内で分けてる
    int idx = blockIdx.y * gridDim.z * threads_per_block + m; // 修正: blockDim.z -> gridDim.z
    if (max_notes < jc + j + a + m || !(jc + j + a + m))
    {
        points[idx] = -1;
        return;
    }
    //points[idx] = memo[(size_t)((jc*1.01f + j + a * 0.5f) * 1000000 / (jc + j + a + m))] + memo[jc] + memo[j] + memo[a] + memo[m];
    //points[idx] =memo[(size_t)((jc*1.01f + j + a * 0.5f) * 1000000 / (jc + j + a + m))];
    points[idx] = ((jc*1.01f + j + a * 0.5f) * 1000000 / (jc + j + a + m));
    return;
}

int main() //これは今後のテスト用に残しておく
{
    int test = 100;
    dim3 grid(1, test+1, (test+1)/threads_per_block + !!(test%threads_per_block));
    dim3 block(threads_per_block);
    typeof_memo *host_memo = new typeof_memo[1010000 + 1], *memo, *points;
    hipMallocManaged(&points, sizeof(typeof_memo) *grid.x * grid.y * grid.z * threads_per_block);
    #pragma omp parallel for
    for (int i = 0; i <= 1010000; i++) {
        auto s_i = to_string(i);
        for (int j = 0; j < s_i.size(); j++) if (s_i[j] == '7') host_memo[i]++;
    }
    hipMalloc(&memo, sizeof(typeof_memo) * (1010000 + 1));
    hipMemcpy(memo, host_memo, sizeof(typeof_memo) * (1010000 + 1), hipMemcpyHostToDevice);
    delete[] host_memo;
    calc_score << <grid, block >> > (3898 - 56 - 18, 56, 40000, memo, points);
    hipDeviceSynchronize();
    for (int a = 0; a < grid.y; a++)
    {
        for (int m = 0; m < grid.z*threads_per_block; m++)
        {
            int idx = a * grid.z * threads_per_block + m;
            if (points[idx] >= 0) {
                if(1007500 <= points[idx]) cout << "(" << a << ", " << m << "): " << points[idx] << endl;
            }
        }
    }
}
